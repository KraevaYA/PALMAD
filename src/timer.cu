#include "timer.cuh"

void start_timer(hipEvent_t* start)
{
    hipEventCreate(start);
    hipEventRecord(*start);
}


float stop_timer(hipEvent_t* start, hipEvent_t* stop)
{
    float milliseconds = 0;
    
    hipEventCreate(stop);
    hipEventRecord(*stop);
    
    hipEventSynchronize(*stop);
    hipEventElapsedTime(&milliseconds, *start, *stop);
    
    return milliseconds;
}
